#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <cstdio>
#include <iomanip>
#include <chrono>
#include <cmath>
#include <thread>
#include <unordered_map>
#include <string>
#include <cassert>
#include <cstring>
#include <vector>

#include "forge_cuda.h"
#include "sha256.cuh"
#include "lamport.h"
#include "signatures.h"

#define SHOW_INTERVAL_MS 2000
#define BLOCK_SIZE 128
#define SHA_PER_ITERATIONS 8388608
#define NUMBLOCKS (SHA_PER_ITERATIONS + BLOCK_SIZE - 1) / BLOCK_SIZE

using std::cout;
using std::endl;
using std::string;
using namespace std::chrono;

// Output string by the device read by host
char *g_out = nullptr;
unsigned char *g_hash_out = nullptr;
int *g_found = nullptr;

static uint64_t nonce = 0;
static uint64_t user_nonce = 0;
static uint64_t last_nonce_since_update = 0;

// Last timestamp we printed debug infos
static std::chrono::high_resolution_clock::time_point t_last_updated;

__device__ bool checkForge(
	unsigned char *sha,
	int *zeroConstraints,
	int *oneConstraints,
	int zeroSize,
	int oneSize)
{
	for (int i = 0; i < zeroSize; i++)
	{
		if ((sha[zeroConstraints[i] / 8] >> (7 - (zeroConstraints[i] % 8)) & 0x01) == 1)
		{
			return false;
		}
	}
	for (int i = 0; i < oneSize; i++)
	{
		if ((sha[oneConstraints[i] / 8] >> (7 - (oneConstraints[i] % 8)) & 0x01) == 0)
		{
			return false;
		}
	}
	return true;
}

// Does the same as sprintf(char*, "%d%s", int, const char*) but a bit faster
__device__ uint8_t nonce_to_str(uint64_t nonce, unsigned char *out)
{
	uint64_t result = nonce;
	uint8_t remainder;
	uint8_t nonce_size = nonce == 0 ? 1 : floor(log10((double)nonce)) + 1;
	uint8_t i = nonce_size;
	while (result >= 10)
	{
		remainder = result % 10;
		result /= 10;
		out[--i] = remainder + '0';
	}

	out[0] = result + '0';
	i = nonce_size;
	out[i] = 0;
	return i;
}

extern __shared__ char array[];
__global__ void forge_kernel(char *out_input_string_nonce,
							 unsigned char *out_found_hash,
							 int *out_found,
							 const char *in_input_string,
							 size_t in_input_string_size,
							 int *zeroConstraints,
							 int *oneConstraints,
							 int zeroSize,
							 int oneSize,
							 uint64_t nonce_offset)
{
	// use shared memory to store string_pre, zero and one constraints and output
	// If this is the first thread of the block, init the input string in shared memory
	// copy the unknown bits after input string
	char *in = (char *)&array[0];
	size_t const minArray0 = static_cast<size_t>(ceil((in_input_string_size) / 8.f) * 8);
	int *zeroConstraints_s = (int *)&array[minArray0];
	int *oneConstraints_s = (int *)&array[minArray0 + 256];
	size_t const minArray = minArray0 + 512;
	uintptr_t sha_addr = threadIdx.x * (64) + minArray;
	uintptr_t nonce_addr = sha_addr + 32;
	unsigned char *sha = (unsigned char *)&array[sha_addr];
	unsigned char *out = (unsigned char *)&array[nonce_addr];

	size_t maxSize = max(oneSize, zeroSize);
	for (size_t tid = threadIdx.x; tid < max(in_input_string_size, maxSize); tid += blockDim.x)
	{
		if (tid < in_input_string_size)
			in[tid] = in_input_string[tid];
		if (tid < zeroSize)
			zeroConstraints_s[tid] = zeroConstraints[tid];
		if (tid < oneSize)
			oneConstraints_s[tid] = oneConstraints[tid];
	}
	__syncthreads(); // Ensure the input string has been written in SMEM

	uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t nonce = idx + nonce_offset;

	// The first byte we can write because there is the input string at the begining
	// Respects the memory padding of 8 bit (char).
	long long trialCnt = 0;
	while (true)
	{
		uint8_t size = nonce_to_str(nonce, out);

		assert(size <= 32);
		{
			SHA256_CTX ctx;
			sha256_init(&ctx);
			sha256_update(&ctx, (unsigned char *)in, in_input_string_size);
			sha256_update(&ctx, out, size);
			sha256_final(&ctx, sha);
		}

		if (checkForge(sha, zeroConstraints_s, oneConstraints_s, zeroSize, oneSize) && atomicExch(out_found, 1) == 0)
		{
			memcpy(out_found_hash, sha, 32);
			memcpy(out_input_string_nonce, in, in_input_string_size);
			memcpy(out_input_string_nonce + in_input_string_size, out, size);
			atomicAdd(out_found, 1);
			break;
		}
		else
		{
			nonce += gridDim.x * blockDim.x;
			trialCnt++;
		}
		if ((trialCnt % 5) == 0 && (*out_found != 0))
			break;
	}
}

void pre_sha256_error_check()
{
	// cuda error check
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

// Prints a 32 bytes sha256 to the hexadecimal form filled with zeroes
void print_hash(const unsigned char *sha256)
{
	for (uint8_t i = 0; i < 32; ++i)
	{
		std::cout << std::hex << std::setfill('0') << std::setw(2) << static_cast<int>(sha256[i]);
	}
	std::cout << std::dec << std::endl;
}

Signature forgeSig(Message forge_msg,
				   int sigIndex[],
				   std::vector<Signature> &sigs,
				   std::vector<Message> &msgs)
{
	Signature sig;
	for (int i = 0; i < 256; i++)
	{
		if ((forge_msg[i / 8] >> (7 - (i % 8)) & 0x01) == 1 && sigIndex[i + 256] != -1)
		{
			sig.Preimage[i] = sigs[sigIndex[i + 256]].Preimage[i];
		}
		else if ((forge_msg[i / 8] >> (7 - (i % 8)) & 0x01) == 0 && sigIndex[i + 256] != -1)
		{
			sig.Preimage[i] = sigs[sigIndex[i]].Preimage[i];
		}
		else
		{
			std::cout << "Won't be able to forge signature with the given message" << std::endl;
			break;
		}
	}
	return sig;
}

// for debug purpose
void print_bits(unsigned char block[])
{
	for (int cur_byte = 0; cur_byte < 32; ++cur_byte)
	{
		unsigned char bk = block[cur_byte];
		for (int j = 7; j >= 0; j--)
		{
			unsigned char mask = 1 << j;
			if (((bk & mask) >> j) == 0)
				std::cout << "0";
			if (((bk & mask) >> j) == 1)
				std::cout << "1";
		}
	}
	std::cout << std::endl;
}

int main()
{
	// Load and check given public key and signatures
	std::unordered_map<std::string, std::string> global_variables = globalVariables();
	PublicKey pub = HexToPublickey(global_variables["hexPubkey1"]);
	Signature sig1 = HexToSignature(global_variables["hexSignature1"]);
	Signature sig2 = HexToSignature(global_variables["hexSignature2"]);
	Signature sig3 = HexToSignature(global_variables["hexSignature3"]);
	Signature sig4 = HexToSignature(global_variables["hexSignature4"]);
	std::vector<Signature> sigs;
	sigs.push_back(sig1);
	sigs.push_back(sig2);
	sigs.push_back(sig3);
	sigs.push_back(sig4);
	Message msg1 = GetMessageFromString(global_variables["msg1_string"]);
	Message msg2 = GetMessageFromString(global_variables["msg2_string"]);
	Message msg3 = GetMessageFromString(global_variables["msg3_string"]);
	Message msg4 = GetMessageFromString(global_variables["msg4_string"]);
	std::vector<Message> msgs;
	msgs.push_back(msg1);
	msgs.push_back(msg2);
	msgs.push_back(msg3);
	msgs.push_back(msg4);

	// verify public keys and signatures
	if (Verify(msg1, pub, sig1))
	{
		std::cout << "sig 1 is verified" << std::endl;
	}
	else
	{
		std::cout << "sig 1 failed to verify" << std::endl;
	}
	if (Verify(msg2, pub, sig2))
	{
		std::cout << "sig 2 is verified" << std::endl;
	}
	else
	{
		std::cout << "sig 2 failed to verify" << std::endl;
	}
	if (Verify(msg3, pub, sig3))
	{
		std::cout << "sig 3 is verified" << std::endl;
	}
	else
	{
		std::cout << "sig 3 failed to verify" << std::endl;
	}
	if (Verify(msg4, pub, sig4))
	{
		std::cout << "sig 4 is verified" << std::endl;
	}
	else
	{
		std::cout << "sig 4 failed to verify" << std::endl;
	}

	// invidividualized message
	std::string msgStringPre = "forge cuda houstonj2013 2024-02-17 ";
	std::cout << "The current message is " << msgStringPre << std::endl;
	std::cout << "Do you want to use new message : (Yes/No)" << std::endl;
	string useNewMsg;
	std::cin >> useNewMsg;
	if (useNewMsg == "Yes" || useNewMsg == "yes")
	{
		std::cout << "Please enter the new message: " << std::endl;
		std::cin >> msgStringPre;
		std::cout << "The new message is : " + msgStringPre << std::endl;
	}
	else
	{
		std::cout << "Still use current message: " << msgStringPre << std::endl;
	}

	std::vector<std::pair<bool, bool>> known_bits(256, {false, false});
	int sigIndex[512] = {-1};
	for (int si = 0; si < msgs.size(); si++)
	{
		Message tempMsg = msgs[si];
		for (int i = 0; i < 256; i++)
		{
			if ((tempMsg[i / 8] >> (7 - (i % 8)) & 0x01) == 1)
			{
				known_bits[i].second = true;
				sigIndex[i + 256] = si;
			}
			else
			{
				known_bits[i].first = true;
				sigIndex[i] = si;
			}
		}
	}
	std::vector<int> zeroConstraints, oneConstraints;
	for (int i = 0; i < 256; i++)
	{
		if (!known_bits[i].first && known_bits[i].second)
			oneConstraints.push_back(i);
		if (known_bits[i].first && !known_bits[i].second)
			zeroConstraints.push_back(i);
	}

	std::cout << "There are " << oneConstraints.size() << " one constraints and " << zeroConstraints.size() << " zero constraints " << std::endl;

	std::string foundMessageString = "forge cuda houstonj2013 2024-02-17 228144973047";

	std::cout << "The saved forge message is " << foundMessageString << std::endl;
	std::cout << "Do you want to use the saved forge: (Yes/No):" << std::endl;
	string useSavedForge;
	std::cin >> useSavedForge;

	if (useSavedForge == "Yes" || useSavedForge == "yes")
	{
		Message foundMsg = GetMessageFromString(foundMessageString);
		std::cout << "The found message is: " << foundMsg.ToHex() << std::endl;
		Signature sig = forgeSig(foundMsg, sigIndex, sigs, msgs);

		if (Verify(foundMsg, pub, sig))
		{
			std::cout << foundMessageString << " was verified to be able to forge signature" << std::endl;
		}
		else
		{
			std::cout << "saved forge " + foundMessageString << " can't be verified" << std::endl;
		}
	}
	else
	{
		hipSetDevice(0);
		hipDeviceSetCacheConfig(hipFuncCachePreferShared);

		t_last_updated = std::chrono::high_resolution_clock::now();

		std::string in = msgStringPre;
		nonce = 1;

		auto start = high_resolution_clock::now();
		const size_t input_size = in.size();

		// Input string for the device
		char *d_in = nullptr;

		// Create the input string for the device
		hipMalloc(&d_in, input_size + 1); // c string has one more length than cpp string
		hipMemcpy(d_in, in.c_str(), input_size + 1, hipMemcpyHostToDevice);

		// create fixed length array for cuda
		int h_zeroConstraints[256] = {-1}, h_oneConstraints[256] = {-1};
		int zeroSize = zeroConstraints.size(), oneSize = oneConstraints.size();
		for (int i = 0; i < zeroSize; i++)
			h_zeroConstraints[i] = zeroConstraints[i];
		for (int i = 0; i < oneSize; i++)
			h_oneConstraints[i] = oneConstraints[i];

		int *d_zeroConstraints = nullptr;
		int *d_oneConstraints = nullptr;
		hipMalloc(&d_zeroConstraints, 256);
		hipMalloc(&d_oneConstraints, 256);
		hipMemcpy(d_zeroConstraints, h_zeroConstraints, 256, hipMemcpyHostToDevice);
		hipMemcpy(d_oneConstraints, h_oneConstraints, 256, hipMemcpyHostToDevice);

		hipMallocManaged(&g_out, input_size + 32 + 1);
		hipMallocManaged(&g_hash_out, 32);
		hipMallocManaged(&g_found, sizeof(int));
		*g_found = 0;

		nonce += user_nonce;
		last_nonce_since_update += user_nonce;

		pre_sha256_error_check();

		size_t dynamic_shared_size = (ceil((input_size + 1) / 8.f) * 8) + (64 * BLOCK_SIZE) + 256 * 2;

		std::cout << "Shared memory is " << dynamic_shared_size / 1024 << "KB" << std::endl;
		std::cout << "Kernel Numblocks: " << NUMBLOCKS << " Block size: " << BLOCK_SIZE << std::endl;

		forge_kernel<<<NUMBLOCKS, BLOCK_SIZE, dynamic_shared_size>>>(g_out,
																	 g_hash_out,
																	 g_found,
																	 d_in,
																	 input_size,
																	 d_zeroConstraints,
																	 d_oneConstraints,
																	 zeroSize,
																	 oneSize,
																	 nonce);

		hipError_t err = hipDeviceSynchronize();
		if (err != hipSuccess)
		{
			throw std::runtime_error("Device error");
		}

		string foundString = g_out;
		Message foundMsg = GetMessageFromString(foundString);
		std::cout << "The found message is: " << foundMsg.ToHex() << std::endl;
		Signature sig = forgeSig(foundMsg, sigIndex, sigs, msgs);

		if (Verify(foundMsg, pub, sig))
		{
			std::cout << foundString << " was verified to be able to forge signature" << std::endl;
		}
		else
		{
			std::cout << "new forged string " + foundString << " can't be verified" << std::endl;
		}

		// if (verifyForge(g_out, known_blocks, pub))
		// {
		// 	std::cout << g_out << " was found to be able to forge signature" << std::endl;
		// }

		hipFree(g_out);
		hipFree(g_hash_out);
		hipFree(g_found);
		hipFree(d_zeroConstraints);
		hipFree(d_oneConstraints);
		hipFree(d_in);

		hipDeviceReset();

		auto stop = high_resolution_clock::now();
		auto duration = duration_cast<milliseconds>(stop - start);
		cout << "program takes " << duration.count() / 1000.f << " seconds to run" << endl;
	}
	return 0;
}
